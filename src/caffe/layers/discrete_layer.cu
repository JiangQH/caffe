#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <string>
#include "caffe/layers/discrete_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {



template <typename Dtype>
__global__ void ForwardLabel(const int nthreads,
	Dtype* const top_data, const Dtype* bottom_data, const int k, const int h, const int w,
	const int discrete_num, bool log_space,
	bool transform, const double delta, const double discrete_min
	) {
	CUDA_KERNEL_LOOP(index, nthreads) {
		const int top_offset = index;
		// get the label
		int lable = 0;
		for (int kk = 0; kk < k; ++kk) {
			const int bottom_offset = top_offset + kk * h * w;
			Dtype value = bottom_data[bottom_offset];
			if (transform) {
				value = (value / 2 + 0.5) * 255;
			}
			if (log_space) {
				value += 1;
				value = log(value);
			}
			int indicate = (value - discrete_min) / delta;
			if (indicate < 0) {
				indicate = 0;
			}
			if (indicate >= discrete_num) {
				indicate = discrete_num - 1;
			}
            lable = lable * discrete_num + indicate;
		}
		top_data[top_offset] = lable;
	}
}



template <typename Dtype>
void DiscreteLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
      
      Dtype* top_data = top[0]->mutable_gpu_data();
      const Dtype* bottom_data = bottom[0]->gpu_data();
      caffe_gpu_set(top[0]->count(), Dtype(0), top_data);
      const int count = bottom[0]->count();

      if (discrete_method_ == "oridnary") {
            bool log_space = false;
            if (discrete_space_ == "log") {
                log_space = true;
            }
            ForwardLabel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, top_data, bottom_data, K_, H_, W_, discrete_num_, log_space, transform_,
            delta_, discrete_min_
            );
      }
      else {
            // todo
      }

}

template <typename Dtype>
void DiscreteLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
        const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
      /** do nothing **/
}

INSTANTIATE_LAYER_GPU_FUNCS(DiscreteLayer);
}// namespace caffe