#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <string>
#include "caffe/layers/discrete_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global_ void ForwardLabel(const int nthreads,
	const Dtype* bottom_data, const int k, const int h, const int w,
	const int discrete_num, string discrete_space,
	bool transform, const double delta, const double discrete_min
	) {
	CUDA_KERNEL_LOOP(index, nthreads) {
		const int top_offset = index;
		// get the label
		int lable = 0;
		for (int kk = 0; kk < k; ++k) {
			const int bottom_offset = top_offset + kk * h * w;
			const Dtype value = bottom_data[bottom_offset];
			if (transform) {
				value = (value / 2 + 0.5) * 255;
			}
			if (discrete_space == "log") {
				value += 1;
				value = log(value);
			}
			int indicate = (value - discrete_min) / delta;
			if (indicate < 0) {
				indicate = 0;
			}
			if (indicate >= discrete_num) {
				indicate = discrete_num - 1;
			}
			label = lable * discrete_num + indicate;

		}
		top_data[top_offset] = lable;
	}
}



template <typename Dtype>
void DiscreteLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
      
      Dtype* top_data = top[0]->mutable_gpu_data();
      const Dtype* bottom_data = bottom[0]->gpu_data();
      caffe_gpu_set(top[0]->count(), Dtype(0), top_data);

      const int count = bottom[0]->count();
      ForwardLable<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      		count, bottom_data, K_, H_, W_, discrete_num_, discrete_space_, transform_,
      		delta_, discrete_min_
      );
}

template <typename Dtype>
void DiscreteLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
      /** do nothing **/
}

}// namespace caffe