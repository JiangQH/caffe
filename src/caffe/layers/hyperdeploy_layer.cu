#include "hip/hip_runtime.h"
#include <vector>
#include <map>
#include <cmath>
#include <algorithm>

#include "caffe/layers/hyperdeploy_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ForwardHypercolumns(const int nthreads,
    const Dtype* bottom_data, const int num, const int bottom_channels,
    const int bottom_height, const int bottom_width, const int sample_pernum,
    const int top_channels, const int top_channel_offset, const int* sampling_list,
    const int original_h, Dtype* const top_data) {
    //forward hypercolumns, separate for each bottom
    CUDA_KERNEL_LOOP(index, nthreads) {
      const int top_n = index / bottom_channels;
      const int bottom_n = top_n / sample_pernum;
      const int bottom_channel = index % bottom_channels; // get the actual channel of the bottom
      const int top_index = bottom_channel + top_n * top_channels + top_channel_offset;
      const int sample_index = sampling_list[top_n];
      const Dtype* const bottom_slice = bottom_data + (bottom_n * bottom_channels + bottom_channel) * bottom_height * bottom_width;
      // get the corresponding bottom_index, according to the top. using bilinear intercept
      const double scale = original_h * 1.0 / bottom_height;
      const int x = sample_index / original_h;
      const int y = sample_index % original_h;
      const double r = x / scale + 1.0 / (2.0 * scale) - 0.5;
      const double c = y / scale + 1.0 / (2.0 * scale) - 0.5;
      const int u = floor(r);
      const int v = floor(c);
      double delta_r = r - u;
      double delta_c = c - v;
      if (u < 0)
        delta_r = 1;
      if (u + 1 >= bottom_width)
        delta_r = 0;
      if (v < 0)
        delta_c = 1;
      if (v + 1 >= bottom_height)
        delta_c = 0;
      // assign the value, notice the boundary check
      double value = 0;
      if ((1 - delta_r) * (1 - delta_c) != 0)
        value += bottom_slice[u * bottom_height + v] * (1 - delta_r) * (1 - delta_c);
      if (delta_r * (1 - delta_c) != 0)
        value += bottom_slice[(u+1) * bottom_height + v] * delta_r * (1 - delta_c);
      if (delta_c * (1 - delta_r) != 0)
        value += bottom_slice[u * bottom_height + v + 1] * delta_c * (1 - delta_r);
      if (delta_r * delta_c != 0)
        value += bottom_slice[(u+1) * bottom_height + v + 1] * delta_r * delta_c;
      top_data[top_index] = value;
    }
}


template <typename Dtype>
void HyperDeployLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
  const vector<Blob<Dtype>*>& top) {
    // generate the sampling list
    selected_points_.clear();
    vector<int> sampling_list;
    for (int n = 0; n < N_; ++n) {
        generate_list(sampling_list, bottom[0], n);
        // for debug usage
        //LOG(INFO) << n <<" sampling points first " << sampling_list[0] << "\n";
        //LOG(INFO) << n << " sampling points last " << sampling_list[sampling_list.size()-1] << "\n";
        selected_points_.insert(selected_points_.end(),
          sampling_list.begin(), sampling_list.end());
    }

    int* cuda_samplelist;
    CUDA_CHECK(hipMalloc(&cuda_samplelist, selected_points_.size() * sizeof(int)));
    CUDA_CHECK(hipMemcpy(cuda_samplelist, &selected_points_[0], selected_points_.size()*sizeof(int), hipMemcpyHostToDevice));
    
    // then forward the hypercolumns
    Dtype* top_hypercolumns = top[0]->mutable_gpu_data();
    int top_channel_offset = 0;
   // const int count0 = top[0]->count();
    //const int top_total_channels = top[0]->shape(1);
    for (int i = 0; i < bottom.size(); ++i) {
      // do it according the corresponding bottom
      const Dtype* bottom_data = bottom[i]->gpu_data();
      const int bottom_channels = bottom[i]->shape(1);
      const int bottom_height = bottom[i]->shape(2);
      const int bottom_width = bottom[i]->shape(3);
      const int nthreads = N_ * sample_num_ * bottom_channels;
      ForwardHypercolumns<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_data, N_, bottom_channels, bottom_height, bottom_width, sample_num_,
        total_channels_, top_channel_offset, cuda_samplelist, H_, top_hypercolumns
      );
      top_channel_offset += bottom_channels;
    }
    CUDA_CHECK(hipFree(cuda_samplelist));
    CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void HyperDeployLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
        const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    // do nothing
}

INSTANTIATE_LAYER_GPU_FUNCS(HyperDeployLayer);
}//namespace caffe
